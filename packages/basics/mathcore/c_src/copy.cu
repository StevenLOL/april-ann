#include "hip/hip_runtime.h"
/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-MLP toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include "cblas_headers.h"
#include "copy.impl.h"
#include "cuda_utils.h"
#include "unused_variable.h"

namespace AprilMath {

#ifdef USE_CUDA
  namespace CUDA {
    
    /***************************************
     ************** CUDA SECTION ***********
     ***************************************/

    template<>
    hipblasStatus_t wrapperCublasCopy<float>(hipblasHandle_t &handle,
                                            int N,
                                            const float *x_mem,
                                            unsigned int x_inc,
                                            float *y_mem,
                                            unsigned int y_inc) {
      return hipblasScopy(handle, N, x_mem, x_inc, y_mem, y_inc);
    }

    template<>
    hipblasStatus_t wrapperCublasCopy<double>(hipblasHandle_t &handle,
                                             int N,
                                             const double *x_mem,
                                             unsigned int x_inc,
                                             double *y_mem,
                                             unsigned int y_inc) {
      return hipblasDcopy(handle, N, x_mem, x_inc, y_mem, y_inc);
    }
    
    template<>
    hipblasStatus_t wrapperCublasCopy<ComplexF>(hipblasHandle_t &handle,
                                               int N,
                                               const ComplexF *x_mem,
                                               unsigned int x_inc,
                                               ComplexF *y_mem,
                                               unsigned int y_inc) {
      return hipblasCcopy(handle, N, reinterpret_cast<const hipComplex*>(x_mem), x_inc,
                         reinterpret_cast<hipComplex*>(y_mem), y_inc);
    }

    template<typename T>
    __global__ void copyLoopKernel(unsigned int N,
                                   const T *x_mem,
                                   unsigned int x_inc,
                                   T *y_mem,
                                   unsigned int y_inc,
                                   unsigned int times,
                                   unsigned int y_ld) {
      unsigned int matrix_x_pos, matrix_y_pos;
      matrix_x_pos = blockIdx.x*blockDim.x + threadIdx.x;
      matrix_y_pos = blockIdx.y*blockDim.y + threadIdx.y;
      if (matrix_x_pos < times && matrix_y_pos < N) {
        unsigned int index_x = matrix_y_pos*x_inc;
        unsigned int index_y = matrix_x_pos*y_ld + matrix_y_pos*y_inc;
        y_mem[index_y] = x_mem[index_x];
      }
    }

  } // namespace CUDA
#endif

  /***************************************
   ************* CBLAS SECTION ***********
   ***************************************/

  template<>
  void wrapperCblasCopy<float>(int N, const float *x_mem, unsigned int x_inc,
                               float *y_mem, unsigned int y_inc) {
    cblas_scopy(N, x_mem, x_inc, y_mem, y_inc);
  }

  template<>
  void wrapperCblasCopy<double>(int N, const double *x_mem, unsigned int x_inc,
                                double *y_mem, unsigned int y_inc) {
    cblas_dcopy(N, x_mem, x_inc, y_mem, y_inc);
  }

  template<>
  void wrapperCblasCopy<ComplexF>(int N, const ComplexF *x_mem, unsigned int x_inc,
                                  ComplexF *y_mem, unsigned int y_inc) {
    cblas_ccopy(N, x_mem, x_inc, y_mem, y_inc);
  }

  /***************************************
   *********** TEMPLATE SECTION **********
   ***************************************/
  
  template<typename T>
  void doCopyBroadcast(int N,
                       GPUMirroredMemoryBlock<T>* x,
                       unsigned int x_inc,
                       GPUMirroredMemoryBlock<T>* A,
                       unsigned int A_inc,
                       unsigned int times,
                       const unsigned int A_stride,
                       bool use_gpu)
  {
    const T *x_mem;
    T *A_mem;
#ifndef USE_CUDA
    UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
    if (use_gpu) {
      //printf("Doing a scopy with comp=1 & cuda=1\n");
      x_mem = x->getGPUForRead();
      A_mem = A->getGPUForWrite();

      const unsigned int MAX_THREADS = CUDA::GPUHelper::getMaxThreadsPerBlock();
      dim3 block, grid;
      // Number of threads on each block dimension
      block.x = min(MAX_THREADS, times);
      block.y = min(MAX_THREADS/block.x, N);
      block.z = 1;

      grid.x = (times/block.x +
                (times % block.x ? 1 : 0));
      grid.y = (N/block.y + (N % block.y ? 1 : 0));
      grid.z = 1;

      CUDA::copyLoopKernel<<<grid, block, 0, CUDA::GPUHelper::getCurrentStream()>>>
        (N, x_mem, x_inc, A_mem, A_inc, times, A_stride);
    }
    else {
      //printf("Doing a scopy with comp=1 & cuda=0\n");
#endif
#ifndef USE_CUDA
      //printf("Doing a scopy with comp=0 & cuda=0\n");
#endif
      x_mem = x->getPPALForRead();
      A_mem = A->getPPALForWrite();

      for (unsigned int i = 0; i < times; i++)
        wrapperCblasCopy(N, 
                         x_mem, x_inc,
                         A_mem + i * A_stride , A_inc);
#ifdef USE_CUDA
    }
#endif
  }

  template void doCopy<int32_t>(int, const GPUMirroredMemoryBlock<int32_t>*,
                                unsigned int,
                                unsigned int,
                                GPUMirroredMemoryBlock<int32_t>*,
                                unsigned int,
                                unsigned int,
                                bool);

  template void doCopy<char>(int, const GPUMirroredMemoryBlock<char>*,
                             unsigned int,
                             unsigned int,
                             GPUMirroredMemoryBlock<char>*,
                             unsigned int,
                             unsigned int,
                             bool);

  template void doCopy<float>(int, const GPUMirroredMemoryBlock<float>*,
                              unsigned int,
                              unsigned int,
                              GPUMirroredMemoryBlock<float>*,
                              unsigned int,
                              unsigned int,
                              bool);

  template void doCopy<double>(int, const GPUMirroredMemoryBlock<double>*,
                               unsigned int,
                               unsigned int,
                               GPUMirroredMemoryBlock<double>*,
                               unsigned int,
                               unsigned int,
                               bool);

  template void doCopy<ComplexF>(int, const GPUMirroredMemoryBlock<ComplexF>*,
                                 unsigned int,
                                 unsigned int,
                                 GPUMirroredMemoryBlock<ComplexF>*,
                                 unsigned int,
                                 unsigned int,
                                 bool);

  template void doCopyBroadcast<float>(int,
                                       GPUMirroredMemoryBlock<float>*,
                                       unsigned int,
                                       GPUMirroredMemoryBlock<float>*,
                                       unsigned int,
                                       unsigned int,
                                       const unsigned int,
                                       bool);

  template void doCopyBroadcast<double>(int,
                                        GPUMirroredMemoryBlock<double>*,
                                        unsigned int,
                                        GPUMirroredMemoryBlock<double>*,
                                        unsigned int,
                                        unsigned int,
                                        const unsigned int,
                                        bool);

  template void doCopyBroadcast<ComplexF>(int,
                                          GPUMirroredMemoryBlock<ComplexF>*,
                                          unsigned int,
                                          GPUMirroredMemoryBlock<ComplexF>*,
                                          unsigned int,
                                          unsigned int,
                                          const unsigned int,
                                          bool);

} // namespace AprilMath
