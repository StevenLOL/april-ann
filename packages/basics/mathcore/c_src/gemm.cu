/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-MLP toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include "wrapper.h"
#include "cuda_utils.h"
#include "unused_variable.h"

#ifdef USE_CUDA
/***************************************
 ************** CUDA SECTION ***********
 ***************************************/

hipblasStatus_t wrapperCublasGemm(hipblasHandle_t &handle,
				 hipblasOperation_t &cublas_a_transpose,
				 hipblasOperation_t &cublas_b_transpose,
				 int m, int n, int k,
				 float *alpha,
				 const float *a_mem,
				 unsigned int a_inc,
				 const float *b_mem,
				 unsigned int b_inc,
				 float *beta,
				 float *c_mem,
				 unsigned int c_inc) {
  return hipblasSgemm(handle, cublas_a_transpose, cublas_b_transpose,
		     m, n, k,
		     alpha, a_mem, a_inc,
		     b_mem, b_inc,
		     beta, c_mem, c_inc);
}

hipblasStatus_t wrapperCublasGemm(hipblasHandle_t &handle,
				 hipblasOperation_t &cublas_a_transpose,
				 hipblasOperation_t &cublas_b_transpose,
				 int m, int n, int k,
				 double *alpha,
				 const double *a_mem,
				 unsigned int a_inc,
				 const double *b_mem,
				 unsigned int b_inc,
				 double *beta,
				 double *c_mem,
				 unsigned int c_inc) {
  return hipblasDgemm(handle, cublas_a_transpose, cublas_b_transpose,
		     m, n, k,
		     alpha, a_mem, a_inc,
		     b_mem, b_inc,
		     beta, c_mem, c_inc);
}

hipblasStatus_t wrapperCublasGemm(hipblasHandle_t &handle,
				 hipblasOperation_t &cublas_a_transpose,
				 hipblasOperation_t &cublas_b_transpose,
				 int m, int n, int k,
				 ComplexF *alpha,
				 const ComplexF *a_mem,
				 unsigned int a_inc,
				 const ComplexF *b_mem,
				 unsigned int b_inc,
				 ComplexF *beta,
				 ComplexF *c_mem,
				 unsigned int c_inc) {
  return hipblasCgemm(handle, cublas_a_transpose, cublas_b_transpose,
		     m, n, k,
		     reinterpret_cast<const hipComplex*>(alpha),
		     reinterpret_cast<const hipComplex*>(a_mem), a_inc,
		     reinterpret_cast<const hipComplex*>(b_mem), b_inc,
		     reinterpret_cast<const hipComplex*>(beta),
                     reinterpret_cast<hipComplex*>(c_mem), c_inc);
}

hipblasStatus_t wrapperCusparseCSRMM(hipsparseHandle_t &handle,
				    hipsparseOperation_t &cusparse_a_transpose,
				    int m, int n, int k,
				    int NNZ,
				    float *alpha,
				    const hipsparseMatDescr_t descrA,
				    const float *a_values_mem,
				    const int *a_first_index_mem,
				    const int *a_indices_mem,
				    const float *b_mem,
				    unsigned int b_inc,
				    float *beta,
				    float *c_mem,
				    unsigned int c_inc) {
  return hipsparseScsrmm(handle, cusparse_a_transpose,
			m, n, k,
			NNZ,
			alpha,
			descrA,
			a_values_mem, a_first_index_mem, a_indices_mem,
			b_mem, b_inc,
			beta, c_mem, c_inc);
}

hipblasStatus_t wrapperCusparseCSRMM(hipsparseHandle_t &handle,
				    hipsparseOperation_t &cusparse_a_transpose,
				    int m, int n, int k,
				    int NNZ,
				    double *alpha,
				    const hipsparseMatDescr_t descrA,
				    const double *a_values_mem,
				    const int *a_first_index_mem,
				    const int *a_indices_mem,
				    const double *b_mem,
				    unsigned int b_inc,
				    double *beta,
				    double *c_mem,
				    unsigned int c_inc) {
  return hipsparseDcsrmm(handle, cusparse_a_transpose,
			m, n, k,
			NNZ,
			alpha,
			descrA,
			a_values_mem, a_first_index_mem, a_indices_mem,
			b_mem, b_inc,
			beta, c_mem, c_inc);
}

hipblasStatus_t wrapperCusparseCSRMM(hipsparseHandle_t &handle,
				    hipsparseOperation_t &cusparse_a_transpose,
				    int m, int n, int k,
				    int NNZ,
				    ComplexF *alpha,
				    const hipsparseMatDescr_t descrA,
				    const ComplexF *a_values_mem,
				    const int *a_first_index_mem,
				    const int *a_indices_mem,
				    const ComplexF *b_mem,
				    unsigned int b_inc,
				    ComplexF *beta,
				    ComplexF *c_mem,
				    unsigned int c_inc) {
  return hipsparseCcsrmm(handle, cusparse_a_transpose,
			m, n, k,
			NNZ,
			reinterpret_cast<const hipComplex*>(alpha),
			descrA,
			reinterpret_cast<const hipComplex*>(a_values_mem),
			a_first_index_mem, a_indices_mem,
			reinterpret_cast<const hipComplex*>(b_mem), b_inc,
			reinterpret_cast<const hipComplex*>(beta),
			reinterpret_cast<const hipComplex*>(c_mem), c_inc);
}

#endif

/***************************************
 ************* CBLAS SECTION ***********
 ***************************************/

void wrapperCblasGemm(CBLAS_ORDER &major_type,
		      CBLAS_TRANSPOSE a_transpose,
		      CBLAS_TRANSPOSE b_transpose,
		      int m, int n, int k,
		      float alpha,
		      const float *a_mem, unsigned int a_inc,
		      const float *b_mem, unsigned int b_inc,
		      float beta, float *c_mem, unsigned int c_inc) {
  cblas_sgemm(major_type, a_transpose, b_transpose,
	      m, n, k,
	      alpha, a_mem, a_inc,
	      b_mem, b_inc,
	      beta, c_mem, c_inc);
}

void wrapperCblasGemm(CBLAS_ORDER &major_type,
		      CBLAS_TRANSPOSE a_transpose,
		      CBLAS_TRANSPOSE b_transpose,
		      int m, int n, int k,
		      double alpha,
		      const double *a_mem, unsigned int a_inc,
		      const double *b_mem, unsigned int b_inc,
		      double beta, double *c_mem, unsigned int c_inc) {
  cblas_dgemm(major_type, a_transpose, b_transpose,
	      m, n, k,
	      alpha, a_mem, a_inc,
	      b_mem, b_inc,
	      beta, c_mem, c_inc);
}

void wrapperCblasGemm(CBLAS_ORDER &major_type,
		      CBLAS_TRANSPOSE a_transpose,
		      CBLAS_TRANSPOSE b_transpose,
		      int m, int n, int k,
		      ComplexF alpha,
		      const ComplexF *a_mem, unsigned int a_inc,
		      const ComplexF *b_mem, unsigned int b_inc,
		      ComplexF beta, ComplexF *c_mem, unsigned int c_inc) {
  cblas_cgemm(major_type, a_transpose, b_transpose,
	      m, n, k,
	      &alpha, a_mem, a_inc,
	      b_mem, b_inc,
	      &beta, c_mem, c_inc);
}

template <typename T>
void wrapperCblasSparseMM(SPARSE_FORMAT sparse_format,
			  CBLAS_TRANSPOSE a_transpose,
			  int m, int n, int k,
			  T alpha,
			  const T *a_values_mem,
			  const int *a_indices_mem,
			  const int *a_first_index_mem,
			  const T *b_mem, unsigned int b_inc,
			  T beta, T *c_mem, unsigned int c_inc) {
  cblas_sparse_mm(sparse_format, a_transpose,
                  m, n, k,
                  alpha, a_values_mem, a_indices_mem, a_first_index_mem,
                  b_mem, static_cast<int>(b_inc),
                  beta, c_mem, static_cast<int>(c_inc));
}

/***************************************
 *********** TEMPLATE SECTION **********
 ***************************************/

template <typename T>
void doGemm(CBLAS_ORDER major_type,
	    CBLAS_TRANSPOSE a_transpose,
	    CBLAS_TRANSPOSE b_transpose,
	    int m,
	    int n,
	    int k,
	    T alpha,
	    const GPUMirroredMemoryBlock<T>* a,
	    unsigned int a_inc,
	    const GPUMirroredMemoryBlock<T>* b,
	    unsigned int b_inc,
	    T beta,
	    GPUMirroredMemoryBlock<T>* c,
	    unsigned int c_inc,
	    unsigned int a_shift,
	    unsigned int b_shift,
	    unsigned int c_shift,
	    bool use_gpu)
{
  const T *a_mem, *b_mem;
  T *c_mem;
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    assert(major_type == CblasColMajor);
    //printf("Doing a sgemm with comp=1 & cuda=1\n");
    a_mem = a->getGPUForRead() + a_shift;
    b_mem = b->getGPUForRead() + b_shift;
    c_mem = c->getGPUForReadAndWrite() + c_shift;
    hipblasOperation_t cublas_a_transpose = getCublasOperation(a_transpose);
    hipblasOperation_t cublas_b_transpose = getCublasOperation(b_transpose);

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = wrapperCublasGemm(handle, cublas_a_transpose, cublas_b_transpose,
			       m, n, k,
			       &alpha, a_mem, a_inc,
			       b_mem, b_inc,
			       &beta, c_mem, c_inc);

    checkCublasError(status);
  }
  else {
    //printf("Doing a sgemm with comp=1 & cuda=0\n");
#endif
    //printf("Doing a sgemm with comp=0 & cuda=0\n");
    a_mem = a->getPPALForRead() + a_shift;
    b_mem = b->getPPALForRead() + b_shift;
    c_mem = c->getPPALForReadAndWrite() + c_shift;

    // matrix matrix product: C = \alpha op(A) op(B) + \beta C
    wrapperCblasGemm(major_type,   // Row or Col Major
		     a_transpose,  // Transpose or not A
		     b_transpose,  // Transpose or not B
		     m,            // num rows of A (before transpose)
		     n,            // num rows at B (before transpose)
		     k,            // Common dimension between A and B
		     alpha,        // Alpha value
		     a_mem,        // A matrix
		     a_inc,        // A matrix stride
		     b_mem,        // B matrix
		     b_inc,        // B matrix stride
		     beta,         // Beta value
		     c_mem,        // C matrix
		     c_inc);       // C matrix stride
#ifdef USE_CUDA
  }
#endif
}

template <typename T>
void doSparseMM(CBLAS_ORDER major_order,
		SPARSE_FORMAT sparse_format,
		CBLAS_TRANSPOSE a_transpose,
		int m,
		int n,
		int k,
		T alpha,
		const GPUMirroredMemoryBlock<T>* a_values,
		const Int32GPUMirroredMemoryBlock* a_indices,
		const Int32GPUMirroredMemoryBlock* a_first_index,
		const GPUMirroredMemoryBlock<T>* b,
		int b_inc,
		T beta,
		GPUMirroredMemoryBlock<T>* c,
		int c_inc,
		int b_shift,
		int c_shift,
		bool use_gpu)
{
  const T *a_values_mem, *b_mem;
  const int *a_indices_mem, *a_first_index_mem;
  T *c_mem;
  const int NNZ = static_cast<int>(a_values->getSize());
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    hipsparseStatus_t status;
    hipsparseHandle_t handle = GPUHelper::getSparseHandler();
    if (major_order != CblasColMajor)
      ERROR_EXIT(128, "Column major matrices are expected\n");
    if (sparse_format != SparseMatrix<T>::CSR_FORMAT)
      a_transpose = NEGATE_CBLAS_TRANSPOSE(a_transpose);
    //printf("Doing a sgemm with comp=1 & cuda=1\n");
    a_values_mem = a_values->getGPUForRead();
    a_indices_mem = a_indices->getGPUForRead();
    a_first_index_mem = a_first_index->getGPUForRead();
    b_mem = b->getGPUForRead() + b_shift;
    c_mem = c->getGPUForReadAndWrite() + c_shift;
    hipsparseOperation_t cusparse_a_transpose = getCusparseOperation(a_transpose);
    
    status = hipsparseSetStream(handle, GPUHelper::getCurrentStream());
    checkCusparseError(status);
    hipsparseMatDescr_t descrA = {
      HIPSPARSE_MATRIX_TYPE_GENERAL,
      0, // fill mode
      0, // diag type
      HIPSPARSE_INDEX_BASE_ZERO
    };
    status = wrapperCusparseCSRMM(handle,
				  cusparse_a_transpose,
				  m, n, k,
				  NNZ,
				  &alpha,
				  descrA,
				  a_values_mem,
				  a_first_index_mem,
				  a_indices_mem,
				  b_mem, b_inc,
				  &beta, c_mem, c_inc);
    checkCusparseError(status);
  }
  else {
    //printf("Doing a sgemm with comp=1 & cuda=0\n");
#endif
    if (major_order != CblasRowMajor)
      ERROR_EXIT(128, "Row major matrices are expected\n");
    //printf("Doing a sgemm with comp=0 & cuda=0\n");
    a_values_mem = a_values->getPPALForRead();
    a_indices_mem = a_indices->getPPALForRead();
    a_first_index_mem = a_first_index->getPPALForRead();
    b_mem = b->getPPALForRead() + b_shift;
    c_mem = c->getPPALForReadAndWrite() + c_shift;
    // matrix matrix product: C = \alpha op(A) op(B) + \beta C
    wrapperCblasSparseMM(sparse_format,
			 a_transpose,
			 m,            // num rows of A (before transpose)
			 n,            // num rows at B (before transpose)
			 k,            // Common dimension between A and B
			 alpha,        // Alpha value
			 a_values_mem,
			 a_indices_mem,
			 a_first_index_mem,
			 b_mem,        // B matrix
			 b_inc,        // B matrix stride
			 beta,         // Beta value
			 c_mem,        // C matrix
			 c_inc);       // C matrix stride
#ifdef USE_CUDA
  }
#endif
}

template void doGemm<float>(CBLAS_ORDER major_type,
			    CBLAS_TRANSPOSE a_transpose,
			    CBLAS_TRANSPOSE b_transpose,
			    int m,
			    int n,
			    int k,
			    float alpha,
			    const GPUMirroredMemoryBlock<float>* a,
			    unsigned int a_inc,
			    const GPUMirroredMemoryBlock<float>* b,
			    unsigned int b_inc,
			    float beta,
			    GPUMirroredMemoryBlock<float>* c,
			    unsigned int c_inc,
			    unsigned int a_shift,
			    unsigned int b_shift,
			    unsigned int c_shift,
			    bool use_gpu);

template void doGemm<double>(CBLAS_ORDER major_type,
			     CBLAS_TRANSPOSE a_transpose,
			     CBLAS_TRANSPOSE b_transpose,
			     int m,
			     int n,
			     int k,
			     double alpha,
			     const GPUMirroredMemoryBlock<double>* a,
			     unsigned int a_inc,
			     const GPUMirroredMemoryBlock<double>* b,
			     unsigned int b_inc,
			     double beta,
			     GPUMirroredMemoryBlock<double>* c,
			     unsigned int c_inc,
			     unsigned int a_shift,
			     unsigned int b_shift,
			     unsigned int c_shift,
			     bool use_gpu);

template void doGemm<ComplexF>(CBLAS_ORDER major_type,
			       CBLAS_TRANSPOSE a_transpose,
			       CBLAS_TRANSPOSE b_transpose,
			       int m,
			       int n,
			       int k,
			       ComplexF alpha,
			       const GPUMirroredMemoryBlock<ComplexF>* a,
			       unsigned int a_inc,
			       const GPUMirroredMemoryBlock<ComplexF>* b,
			       unsigned int b_inc,
			       ComplexF beta,
			       GPUMirroredMemoryBlock<ComplexF>* c,
			       unsigned int c_inc,
			       unsigned int a_shift,
			       unsigned int b_shift,
			       unsigned int c_shift,
			       bool use_gpu);

template void doSparseMM<float>(CBLAS_ORDER major_order,
                                SPARSE_FORMAT sparse_format,
                                CBLAS_TRANSPOSE a_transpose,
                                int m,
                                int n,
                                int k,
                                float alpha,
                                const GPUMirroredMemoryBlock<float>* a_values,
                                const Int32GPUMirroredMemoryBlock* a_indices,
                                const Int32GPUMirroredMemoryBlock* a_first_index,
                                const GPUMirroredMemoryBlock<float>* b,
                                int b_inc,
                                float beta,
                                GPUMirroredMemoryBlock<float>* c,
                                int c_inc,
                                int b_shift,
                                int c_shift,
                                bool use_gpu);

template void doSparseMM<double>(CBLAS_ORDER major_order,
                                 SPARSE_FORMAT sparse_format,
                                 CBLAS_TRANSPOSE a_transpose,
                                 int m,
                                 int n,
                                 int k,
                                 double alpha,
                                 const GPUMirroredMemoryBlock<double>* a_values,
                                 const Int32GPUMirroredMemoryBlock* a_indices,
                                 const Int32GPUMirroredMemoryBlock* a_first_index,
                                 const GPUMirroredMemoryBlock<double>* b,
                                 int b_inc,
                                 double beta,
                                 GPUMirroredMemoryBlock<double>* c,
                                 int c_inc,
                                 int b_shift,
                                 int c_shift,
                                 bool use_gpu);

template void doSparseMM<ComplexF>(CBLAS_ORDER major_order,
                                   SPARSE_FORMAT sparse_format,
                                   CBLAS_TRANSPOSE a_transpose,
                                   int m,
                                   int n,
                                   int k,
                                   ComplexF alpha,
                                   const GPUMirroredMemoryBlock<ComplexF>* a_values,
                                   const Int32GPUMirroredMemoryBlock* a_indices,
                                   const Int32GPUMirroredMemoryBlock* a_first_index,
                                   const GPUMirroredMemoryBlock<ComplexF>* b,
                                   int b_inc,
                                   ComplexF beta,
                                   GPUMirroredMemoryBlock<ComplexF>* c,
                                   int c_inc,
                                   int b_shift,
                                   int c_shift,
                                   bool use_gpu);
