/*
 * This file is part of the Neural Network modules of the APRIL toolkit (A
 * Pattern Recognizer In Lua).
 *
 * Copyright 2012, Salvador España-Boquera, Adrian Palacios Corella, Francisco
 * Zamora-Martinez
 *
 * The APRIL-MLP toolkit is free software; you can redistribute it and/or modify it
 * under the terms of the GNU General Public License version 3 as
 * published by the Free Software Foundation
 *
 * This library is distributed in the hope that it will be useful, but WITHOUT
 * ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 * FITNESS FOR A PARTICULAR PURPOSE. See the GNU General Public License
 * for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this library; if not, write to the Free Software Foundation,
 * Inc., 59 Temple Place, Suite 330, Boston, MA 02111-1307 USA
 *
 */
#include "wrapper.h"
#include "cuda_utils.h"
#include "unused_variable.h"

#ifdef USE_CUDA
/***************************************
 ************** CUDA SECTION ***********
 ***************************************/

hipblasStatus_t wrapperCublasGemv(hipblasHandle_t &handle,
				 hipblasOperation_t &cublas_a_transpose,
				 int m, int n,
				 const float *alpha,
				 const float *a_mem,
				 unsigned int a_inc,
				 const float *x_mem,
				 unsigned int x_inc,
				 const float *beta,
				 float *y_mem,
				 unsigned int y_inc) {
  return hipblasSgemv(handle, cublas_a_transpose,
		     m, n,
		     alpha, a_mem, a_inc,
		     x_mem, x_inc,
		     beta, y_mem, y_inc);
}

hipblasStatus_t wrapperCublasGemv(hipblasHandle_t &handle,
				 hipblasOperation_t &cublas_a_transpose,
				 int m, int n,
				 const double *alpha,
				 const double *a_mem,
				 unsigned int a_inc,
				 const double *x_mem,
				 unsigned int x_inc,
				 const double *beta,
				 double *y_mem,
				 unsigned int y_inc) {
  return hipblasSgemv(handle, cublas_a_transpose,
		     m, n,
		     alpha, a_mem, a_inc,
		     x_mem, x_inc,
		     beta, y_mem, y_inc);
}

hipblasStatus_t wrapperCublasGemv(hipblasHandle_t &handle,
				 hipblasOperation_t &cublas_a_transpose,
				 int m, int n,
				 const ComplexF *alpha,
				 const ComplexF *a_mem,
				 unsigned int a_inc,
				 const ComplexF *x_mem,
				 unsigned int x_inc,
				 const ComplexF *beta,
				 ComplexF *y_mem,
				 unsigned int y_inc) {
  return hipblasCgemv(handle, cublas_a_transpose,
		     m, n,
		     reinterpret_cast<const hipComplex*>(alpha),
                     reinterpret_cast<const hipComplex*>(a_mem), a_inc,
		     reinterpret_cast<const hipComplex*>(x_mem), x_inc,
		     reinterpret_cast<const hipComplex*>(beta),
                     reinterpret_cast<hipComplex*>(y_mem), y_inc);
}

hipsparseStatus_t wrapperCusparseCSRGemv(hipsparseHandle_t &handle,
                                        hipsparseOperation_t &cusparse_a_transpose,
                                        int m, int n, int NNZ,
                                        const float *alpha,
                                        hipsparseMatDescr_t descrA,
                                        const float *a_values_mem,
                                        const int *a_indices_mem,
                                        const int *a_first_index_mem,
                                        const float *x_mem,
                                        unsigned int x_inc,
                                        const float *beta,
                                        float *y_mem,
                                        unsigned int y_inc) {
  if (x_inc != 1 || y_inc != 1)
    ERROR_EXIT(128, "Not implemented for non contiguous vectors\n");
  return hipsparseScsrmv(handle, cusparse_a_transpose,
                        m, n, NNZ,
                        alpha,
                        descrA,
                        a_values_mem,
                        a_first_index_mem,
                        a_indices_mem
                        x_mem, x_inc,
                        beta, y_mem, y_inc);
}

hipsparseStatus_t wrapperCusparseCSRGemv(hipsparseHandle_t &handle,
                                        hipsparseOperation_t &cusparse_a_transpose,
                                        int m, int n, int NNZ,
                                        const double *alpha,
                                        hipsparseMatDescr_t descrA,
                                        const double *a_values_mem,
                                        const int *a_indices_mem,
                                        const int *a_first_index_mem,
                                        const double *x_mem,
                                        unsigned int x_inc,
                                        const double *beta,
                                        double *y_mem,
                                        unsigned int y_inc) {
  if (x_inc != 1 || y_inc != 1)
    ERROR_EXIT(128, "Not implemented for non contiguous vectors\n");
  return hipsparseScsrmv(handle, cusparse_a_transpose,
                        m, n, NNZ,
                        alpha,
                        descrA,
                        a_values_mem,
                        a_first_index_mem,
                        a_indices_mem
                        x_mem, x_inc,
                        beta, y_mem, y_inc);
}

hipsparseStatus_t wrapperCusparseCSRGemv(hipsparseHandle_t &handle,
                                        hipsparseOperation_t &cusparse_a_transpose,
                                        int m, int n, int NNZ,
                                        const ComplexF *alpha,
                                        hipsparseMatDescr_t descrA,
                                        const ComplexF *a_values_mem,
                                        const int *a_indices_mem,
                                        const int *a_first_index_mem,
                                        const ComplexF *x_mem,
                                        unsigned int x_inc,
                                        const ComplexF *beta,
                                        ComplexF *y_mem,
                                        unsigned int y_inc) {
  if (x_inc != 1 || y_inc != 1)
    ERROR_EXIT(128, "Not implemented for non contiguous vectors\n");
  return hipsparseScsrmv(handle, cusparse_a_transpose,
                        m, n, NNZ,
                        alpha,
                        descrA,
                        a_values_mem,
                        a_first_index_mem,
                        a_indices_mem
                        x_mem, x_inc,
                        beta, y_mem, y_inc);
}
#endif

/***************************************
 ************* CBLAS SECTION ***********
 ***************************************/

void wrapperCblasGemv(CBLAS_ORDER &major_order,
		      CBLAS_TRANSPOSE a_transpose,
		      int m, int n,
		      float alpha,
		      const float *a_mem, unsigned int a_inc,
		      const float *x_mem, unsigned int x_inc,
		      float beta,
		      float *y_mem, unsigned int y_inc) {
  cblas_sgemv(major_order, a_transpose, m, n, alpha, a_mem, a_inc,
	      x_mem, x_inc, beta, y_mem, y_inc);
}

void wrapperCblasGemv(CBLAS_ORDER &major_order,
		      CBLAS_TRANSPOSE a_transpose,
		      int m, int n,
		      ComplexF alpha,
		      const ComplexF *a_mem, unsigned int a_inc,
		      const ComplexF *x_mem, unsigned int x_inc,
		      ComplexF beta,
		      ComplexF *y_mem, unsigned int y_inc) {
  cblas_cgemv(major_order, a_transpose, m, n, &alpha, a_mem, a_inc,
	      x_mem, x_inc, &beta, y_mem, y_inc);
}

template <typename T>
void wrapperCblasSparseMM(SPARSE_FORMAT sparse_format,
			  CBLAS_TRANSPOSE a_transpose,
			  int m, int n,
			  T alpha,
			  const T *a_values_mem,
			  const int *a_indices_mem,
			  const int *a_first_index_mem,
			  const T *x_mem, unsigned int x_inc,
			  T beta, T *y_mem, unsigned int y_inc) {
  cblas_sparse_mv(sparse_format, a_transpose,
                  m, n,
                  alpha, a_values_mem, a_indices_mem, a_first_index_mem,
                  x_mem, static_cast<int>(x_inc),
                  beta, y_mem, static_cast<int>(y_inc));
}

/***************************************
 *********** TEMPLATE SECTION **********
 ***************************************/

template<typename T>
void doGemv(CBLAS_ORDER major_order, CBLAS_TRANSPOSE a_transpose,
	    int m, int n,
	    T alpha, GPUMirroredMemoryBlock<T> *a, unsigned int a_inc,
	    GPUMirroredMemoryBlock<T> *x, unsigned int x_inc,
	    T beta, GPUMirroredMemoryBlock<T> *y, unsigned int y_inc,
	    unsigned int a_shift, unsigned int x_shift, unsigned int y_shift,
	    bool use_gpu) {
  const T *a_mem, *x_mem;
  T *y_mem;
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    hipblasStatus_t status;
    hipblasHandle_t handle = GPUHelper::getHandler();
    assert(major_order == CblasColMajor);
    hipblasOperation_t cublas_a_transpose = getCublasOperation(a_transpose);
    a_mem = a->getGPUForRead() + a_shift;
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForReadAndWrite() + y_shift;

    status = hipblasSetStream(handle, GPUHelper::getCurrentStream());
    checkCublasError(status);

    status = wrapperCublasGemv(handle, cublas_a_transpose,
			       m, n,
			       &alpha, a_mem, a_inc,
			       x_mem, x_inc,
			       &beta, y_mem, y_inc);
    
    checkCublasError(status);
  }
  else {
#endif
    a_mem = a->getPPALForRead() + a_shift;
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForReadAndWrite() + y_shift;
    wrapperCblasGemv(major_order, a_transpose,
		     m, n,
		     alpha, a_mem, a_inc,
		     x_mem, x_inc,
		     beta, y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
}

template<typename T>
void doSparseGemv(CBLAS_ORDER major_order, SPARSE_FORMAT sparse_format,
                  CBLAS_TRANSPOSE a_transpose,
                  int m, int n,
                  T alpha,
                  GPUMirroredMemoryBlock<T> *a_values,
                  Int32GPUMirroredMemoryBlock *a_indices,
                  Int32GPUMirroredMemoryBlock *a_first_index,
                  GPUMirroredMemoryBlock<T> *x, unsigned int x_inc,
                  T beta, GPUMirroredMemoryBlock<T> *y, unsigned int y_inc,
                  unsigned int x_shift, unsigned int y_shift,
                  bool use_gpu) {
  const T *a_values_mem, *x_mem;
  const int *a_indices_mem, *a_first_index_mem;
  T *y_mem;
  const int NNZ = static_cast<int>(a_values->getSize());
#ifndef USE_CUDA
  UNUSED_VARIABLE(use_gpu);
#endif
#ifdef USE_CUDA
  if (use_gpu) {
    hipsparseStatus_t status;
    hipsparseHandle_t handle = GPUHelper::getSparseHandler();
    if (major_order != CblasColMajor)
      ERROR_EXIT(128, "Column major matrices are expected\n");
    if (sparse_format != SparseMatrix<T>::CSR_FORMAT)
      a_transpose = NEGATE_CBLAS_TRANSPOSE(a_transpose);
    a_values_mem = a_values->getGPUForRead();
    a_indices_mem = a_indices->getGPUForRead();
    a_first_index_mem = a_first_index->getGPUForRead();
    x_mem = x->getGPUForRead() + x_shift;
    y_mem = y->getGPUForReadAndWrite() + y_shift;
    hipsparseOperation_t cusparse_a_transpose = getCusparseOperation(a_transpose);
    
    status = hipsparseSetStream(handle, GPUHelper::getCurrentStream());
    checkCusparseError(status);
    hipsparseMatDescr_t descrA = {
      HIPSPARSE_MATRIX_TYPE_GENERAL,
      0, // fill mode
      0, // diag type
      HIPSPARSE_INDEX_BASE_ZERO
    };
    
    status = wrapperCusparseCSRGemv(handle,
                                    cusparse_a_transpose,
                                    m, n, NNZ,
                                    &alpha,
                                    descrA,
                                    a_values_mem,
                                    a_first_index_mem,
                                    a_indices_mem,
                                    x_mem, x_inc,
                                    &beta, y_mem, y_inc);
    checkCusparseError(status);

  }
  else {
#endif
    if (major_order != CblasRowMajor)
      ERROR_EXIT(128, "Row major matrices are expected\n");
    a_values_mem = a_values->getPPALForRead();
    a_indices_mem = a_indices->getPPALForRead();
    a_first_index_mem = a_first_index->getPPALForRead();
    x_mem = x->getPPALForRead() + x_shift;
    y_mem = y->getPPALForReadAndWrite() + y_shift;
    wrapperCblasSparseGemv(sparse_format,
                           a_transpose,
                           m, n,
                           alpha,
                           a_values_mem,
                           a_indices_mem,
                           a_first_index_mem,
                           x_mem, x_inc,
                           beta, y_mem, y_inc);
#ifdef USE_CUDA
  }
#endif
}

template void doGemv<float>(CBLAS_ORDER major_order, CBLAS_TRANSPOSE a_transpose,
			    int m, int n,
			    float alpha, GPUMirroredMemoryBlock<float> *a, unsigned int a_inc,
			    GPUMirroredMemoryBlock<float> *x, unsigned int x_inc,
			    float beta, GPUMirroredMemoryBlock<float> *y, unsigned int y_inc,
			    unsigned int a_shift, unsigned int x_shift, unsigned int y_shift,
			    bool use_gpu);

template void doGemv<double>(CBLAS_ORDER major_order, CBLAS_TRANSPOSE a_transpose,
                             int m, int n,
                             double alpha, GPUMirroredMemoryBlock<double> *a, unsigned int a_inc,
                             GPUMirroredMemoryBlock<double> *x, unsigned int x_inc,
                             double beta, GPUMirroredMemoryBlock<double> *y, unsigned int y_inc,
                             unsigned int a_shift, unsigned int x_shift, unsigned int y_shift,
                             bool use_gpu);

template void doGemv<ComplexF>(CBLAS_ORDER major_order, CBLAS_TRANSPOSE a_transpose,
			       int m, int n,
			       ComplexF alpha, GPUMirroredMemoryBlock<ComplexF> *a, unsigned int a_inc,
			       GPUMirroredMemoryBlock<ComplexF> *x, unsigned int x_inc,
			       ComplexF beta, GPUMirroredMemoryBlock<ComplexF> *y, unsigned int y_inc,
			       unsigned int a_shift, unsigned int x_shift, unsigned int y_shift,
			       bool use_gpu);

template void doSparseGemv<float>(CBLAS_ORDER major_order,
                                  SPARSE_FORMAT sparse_format,
                                  CBLAS_TRANSPOSE a_transpose,
                                  int m, int n,
                                  float alpha,
                                  GPUMirroredMemoryBlock<float> *a_values,
                                  Int32GPUMirroredMemoryBlock *a_indices,
                                  Int32GPUMirroredMemoryBlock *a_first_index,
                                  GPUMirroredMemoryBlock<float> *x, unsigned int x_inc,
                                  float beta, GPUMirroredMemoryBlock<float> *y, unsigned int y_inc,
                                  unsigned int x_shift, unsigned int y_shift,
                                  bool use_gpu);

template void doSparseGemv<double>(CBLAS_ORDER major_order,
                                   SPARSE_FORMAT sparse_format,
                                   CBLAS_TRANSPOSE a_transpose,
                                   int m, int n,
                                   double alpha,
                                   GPUMirroredMemoryBlock<double> *a_values,
                                   Int32GPUMirroredMemoryBlock *a_indices,
                                   Int32GPUMirroredMemoryBlock *a_first_index,
                                   GPUMirroredMemoryBlock<double> *x, unsigned int x_inc,
                                   double beta, GPUMirroredMemoryBlock<double> *y, unsigned int y_inc,
                                   unsigned int x_shift, unsigned int y_shift,
                                   bool use_gpu);

template void doSparseGemv<ComplexF>(CBLAS_ORDER major_order,
                                     SPARSE_FORMAT sparse_format,
                                     CBLAS_TRANSPOSE a_transpose,
                                     int m, int n,
                                     ComplexF alpha,
                                     GPUMirroredMemoryBlock<ComplexF> *a_values,
                                     Int32GPUMirroredMemoryBlock *a_indices,
                                     Int32GPUMirroredMemoryBlock *a_first_index,
                                     GPUMirroredMemoryBlock<ComplexF> *x, unsigned int x_inc,
                                     ComplexF beta, GPUMirroredMemoryBlock<ComplexF> *y, unsigned int y_inc,
                                     unsigned int x_shift, unsigned int y_shift,
                                     bool use_gpu);
